#include "hip/hip_runtime.h"
#include <cub/block/block_load.cuh>
#include <cub/block/block_radix_sort.cuh>
#include <cub/block/block_store.cuh>

#include "device.h"
#include "gpu_cuda.h"
#include "paddle/extension.h"
#include "prod_virial.h"

#define CHECK_INPUT(x) PD_CHECK(x.is_gpu(), #x " must be a GPU Tensor.")
#define CHECK_INPUT_ON_CPU(x) PD_CHECK(x.is_cpu(), #x " must be a CPU Tensor.")
#define CHECK_INPUT_DIM(x, value) \
  PD_CHECK(x.shape().size() == value, #x "'s dim should be " #value ".")
#define CHECK_INPUT_READY(x) \
  PD_CHECK(x.initialized(), #x " must be initialized before usage.")

template <typename FPTYPE, int THREADS_PER_BLOCK>
__global__ void atom_virial_reduction(FPTYPE* virial,
                                      const FPTYPE* atom_virial,
                                      const int nall) {
  unsigned int bid = blockIdx.x;
  unsigned int tid = threadIdx.x;
  __shared__ FPTYPE data[THREADS_PER_BLOCK];
  data[tid] = (FPTYPE)0.;
  for (int ii = tid; ii < nall; ii += THREADS_PER_BLOCK) {
    data[tid] += atom_virial[ii * 9 + bid];
  }
  __syncthreads();
  // do reduction in shared memory
  for (int ii = THREADS_PER_BLOCK >> 1; ii > 0; ii >>= 1) {
    if (tid < ii) {
      data[tid] += data[tid + ii];
    }
    __syncthreads();
  }
  // write result for this block to global memory
  if (tid == 0) virial[bid] = data[0];
}

template <typename FPTYPE>
__global__ void virial_deriv_wrt_neighbors_a(FPTYPE* virial,
                                             FPTYPE* atom_virial,
                                             const FPTYPE* net_deriv,
                                             const FPTYPE* in_deriv,
                                             const FPTYPE* rij,
                                             const int* nlist,
                                             const int nloc,
                                             const int nnei) {
  // idx -> nloc
  // idy -> nnei
  // idz = dd0 * 3 + dd1
  // dd0 = idz / 3
  // dd1 = idz % 3
  const int_64 idx = blockIdx.x;
  const unsigned int idy = blockIdx.y * blockDim.x + threadIdx.x;
  const unsigned int idz = threadIdx.y;
  const int ndescrpt = nnei * 4;
  if (idy >= nnei) {
    return;
  }
  int j_idx = nlist[idx * nnei + idy];
  if (j_idx < 0) {
    return;
  }
  // atomicAdd(
  //    virial + idz,
  //    net_deriv[idx * ndescrpt + idy * 4 + idw] * rij[idx * nnei * 3 + idy * 3
  //    + idz / 3] * in_deriv[idx * ndescrpt * 3 + (idy * 4 + idw) * 3 + idz %
  //    3]);
  FPTYPE virial_tmp = (FPTYPE)0.;
  for (int idw = 0; idw < 4; ++idw) {
    virial_tmp += net_deriv[idx * ndescrpt + idy * 4 + idw] *
                  rij[idx * nnei * 3 + idy * 3 + idz % 3] *
                  in_deriv[idx * ndescrpt * 3 + (idy * 4 + idw) * 3 + idz / 3];
  }
  atomicAdd(atom_virial + j_idx * 9 + idz, virial_tmp);
}

template <typename FPTYPE>
__global__ void virial_deriv_wrt_neighbors_r(FPTYPE* virial,
                                             FPTYPE* atom_virial,
                                             const FPTYPE* net_deriv,
                                             const FPTYPE* in_deriv,
                                             const FPTYPE* rij,
                                             const int* nlist,
                                             const int nloc,
                                             const int nnei) {
  // idx -> nloc
  // idy -> nnei
  // idz = dd0 * 3 + dd1
  // dd0 = idz / 3
  // dd1 = idz % 3
  const int_64 idx = blockIdx.x;
  const unsigned int idy = blockIdx.y * blockDim.x + threadIdx.x;
  const unsigned int idz = threadIdx.y;
  const int ndescrpt = nnei * 1;

  if (idy >= nnei) {
    return;
  }
  int j_idx = nlist[idx * nnei + idy];
  if (j_idx < 0) {
    return;
  }
  // atomicAdd(
  //    virial + idz,
  //    net_deriv[idx * ndescrpt + idy * 4 + idw] * rij[idx * nnei * 3 + idy * 3
  //    + idz / 3] * in_deriv[idx * ndescrpt * 3 + (idy * 4 + idw) * 3 + idz %
  //    3]);
  atomicAdd(atom_virial + j_idx * 9 + idz,
            net_deriv[idx * ndescrpt + idy] *
                rij[idx * nnei * 3 + idy * 3 + idz % 3] *
                in_deriv[idx * ndescrpt * 3 + idy * 3 + idz / 3]);
}

namespace deepmd {
template <typename FPTYPE>
void prod_virial_a_gpu_cuda(FPTYPE* virial,
                            FPTYPE* atom_virial,
                            const FPTYPE* net_deriv,
                            const FPTYPE* in_deriv,
                            const FPTYPE* rij,
                            const int* nlist,
                            const int nloc,
                            const int nall,
                            const int nnei) {
  DPErrcheck(hipMemset(virial, 0, sizeof(FPTYPE) * 9));
  DPErrcheck(hipMemset(atom_virial, 0, sizeof(FPTYPE) * 9 * nall));

  const int LEN = 16;
  int nblock = (nnei + LEN - 1) / LEN;
  dim3 block_grid(nloc, nblock);
  dim3 thread_grid(LEN, 9);
  // compute virial of a frame
  virial_deriv_wrt_neighbors_a<<<block_grid, thread_grid>>>(
      virial, atom_virial, net_deriv, in_deriv, rij, nlist, nloc, nnei);
  DPErrcheck(hipGetLastError());
  DPErrcheck(hipDeviceSynchronize());
  // reduction atom_virial to virial
  atom_virial_reduction<FPTYPE, TPB><<<9, TPB>>>(virial, atom_virial, nall);
  DPErrcheck(hipGetLastError());
  DPErrcheck(hipDeviceSynchronize());
}

template <typename FPTYPE>
void prod_virial_r_gpu_cuda(FPTYPE* virial,
                            FPTYPE* atom_virial,
                            const FPTYPE* net_deriv,
                            const FPTYPE* in_deriv,
                            const FPTYPE* rij,
                            const int* nlist,
                            const int nloc,
                            const int nall,
                            const int nnei) {
  DPErrcheck(hipMemset(virial, 0, sizeof(FPTYPE) * 9));
  DPErrcheck(hipMemset(atom_virial, 0, sizeof(FPTYPE) * 9 * nall));

  const int LEN = 16;
  int nblock = (nnei + LEN - 1) / LEN;
  dim3 block_grid(nloc, nblock);
  dim3 thread_grid(LEN, 9);
  // compute virial of a frame
  virial_deriv_wrt_neighbors_r<<<block_grid, thread_grid>>>(
      virial, atom_virial, net_deriv, in_deriv, rij, nlist, nloc, nnei);
  DPErrcheck(hipGetLastError());
  DPErrcheck(hipDeviceSynchronize());
  // reduction atom_virial to virial
  atom_virial_reduction<FPTYPE, TPB><<<9, TPB>>>(virial, atom_virial, nall);
  DPErrcheck(hipGetLastError());
  DPErrcheck(hipDeviceSynchronize());
}
}  // namespace deepmd

template <typename data_t>
void ProdVirialSeAOpForwardCUDAKernel(int nloc,
                                      int nall,
                                      int ndescrpt,
                                      int nnei,
                                      int nframes,
                                      data_t* p_virial,
                                      data_t* p_atom_virial,
                                      const data_t* p_net_deriv,
                                      const data_t* p_in_deriv,
                                      const data_t* p_rij,
                                      const int* p_nlist) {
  for (int kk = 0; kk < nframes; ++kk) {
    data_t* virial = p_virial + kk * 9;
    data_t* atom_virial = p_atom_virial + kk * nall * 9;
    const data_t* net_deriv = p_net_deriv + kk * nloc * ndescrpt;
    const data_t* in_deriv = p_in_deriv + kk * nloc * ndescrpt * 3;
    const data_t* rij = p_rij + kk * nloc * nnei * 3;
    const int* nlist = p_nlist + kk * nloc * nnei;
    deepmd::prod_virial_a_gpu_cuda(virial, atom_virial, net_deriv, in_deriv,
                                   rij, nlist, nloc, nall, nnei);
  }
}

std::vector<paddle::Tensor> ProdVirialSeAOpCUDAForward(
    const paddle::Tensor& net_deriv_tensor,
    const paddle::Tensor& in_deriv_tensor,
    const paddle::Tensor& rij_tensor,
    const paddle::Tensor& nlist_tensor,
    const paddle::Tensor& natoms_tensor,
    int n_a_sel,
    int n_r_sel) {
  CHECK_INPUT(net_deriv_tensor);
  CHECK_INPUT(in_deriv_tensor);
  CHECK_INPUT(rij_tensor);
  CHECK_INPUT(nlist_tensor);
  CHECK_INPUT_ON_CPU(natoms_tensor);  // TODO:
  // 暂时指定python端必须为cpu，gpu的copy_to会导致返回的指针数据不对

  CHECK_INPUT_DIM(net_deriv_tensor, 2);
  CHECK_INPUT_DIM(in_deriv_tensor, 2);
  CHECK_INPUT_DIM(rij_tensor, 2);
  CHECK_INPUT_DIM(nlist_tensor, 2);
  CHECK_INPUT_DIM(natoms_tensor, 1);

  PD_CHECK(natoms_tensor.shape()[0] >= 3,
           "number of atoms should be larger than (or equal to) 3");
  const int* natoms = natoms_tensor.data<int>();
  int nloc = natoms[0];
  int nall = natoms[1];
  int nnei = nlist_tensor.shape()[1] / nloc;
  int nframes = net_deriv_tensor.shape()[0];
  int ndescrpt = net_deriv_tensor.shape()[1] / nloc;
  PD_CHECK(nframes == in_deriv_tensor.shape()[0],
           "number of samples should match");
  PD_CHECK(nframes == rij_tensor.shape()[0], "number of samples should match");
  PD_CHECK(nframes == nlist_tensor.shape()[0],
           "number of samples should match");
  PD_CHECK(nloc * ndescrpt * 3 == in_deriv_tensor.shape()[1],
           "number of descriptors should match");
  PD_CHECK((nloc * nnei * 3) == rij_tensor.shape()[1],
           "dim of rij should be nnei * 3");

  std::vector<int64_t> virial_shape{nframes, 9};
  std::vector<int64_t> atom_virial_shape{nframes, 9 * nall};
  paddle::Tensor virial_tensor =
      paddle::Tensor(paddle::PlaceType::kGPU, virial_shape);
  paddle::Tensor atom_virial_tensor =
      paddle::Tensor(paddle::PlaceType::kGPU, atom_virial_shape);

  PD_DISPATCH_FLOATING_TYPES(
      net_deriv_tensor.type(), "prod_virial_se_a_cuda_forward_kernel", ([&] {
        ProdVirialSeAOpForwardCUDAKernel<data_t>(
            nloc, nall, ndescrpt, nnei, nframes,
            virial_tensor.mutable_data<data_t>(),
            atom_virial_tensor.mutable_data<data_t>(),
            net_deriv_tensor.data<data_t>(), in_deriv_tensor.data<data_t>(),
            rij_tensor.data<data_t>(), nlist_tensor.data<int>());
      }));

  return {virial_tensor, atom_virial_tensor};
}
