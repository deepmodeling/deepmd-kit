#include "hip/hip_runtime.h"
#include <vector>
#include <climits>
#include <stdio.h>
#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>
#include <cub/block/block_radix_sort.cuh>
#include <hip/hip_runtime.h>

#define cudaErrcheck(res) {cudaAssert((res), __FILE__, __LINE__);}
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"cuda assert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

typedef unsigned long long int_64;

template <
    typename    Key,
    int         BLOCK_THREADS,
    int         ITEMS_PER_THREAD>
__launch_bounds__ (BLOCK_THREADS)
__global__ void BlockSortKernel(
    Key * d_in,
    Key * d_out)            // Tile of output
{   
    enum { TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD };
    // Specialize BlockLoad type for our thread block (uses warp-striped loads for coalescing, then transposes in shared memory to a blocked arrangement)
    typedef hipcub::BlockLoad<Key, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> BlockLoadT;
    // Specialize BlockRadixSort type for our thread block
    typedef hipcub::BlockRadixSort<Key, BLOCK_THREADS, ITEMS_PER_THREAD> BlockRadixSortT;
    // Shared memory
    __shared__ union TempStorage
    {
        typename BlockLoadT::TempStorage        load;
        typename BlockRadixSortT::TempStorage   sort;
    } temp_storage;
    // Per-thread tile items
    Key items[ITEMS_PER_THREAD];
    // Our current block's offset
    int block_offset = blockIdx.x * TILE_SIZE;
    // Load items into a blocked arrangement
    BlockLoadT(temp_storage.load).Load(d_in + block_offset, items);
    // Barrier for smem reuse
    __syncthreads();
    // Sort keys
    BlockRadixSortT(temp_storage.sort).SortBlockedToStriped(items);
    // Store output in striped fashion
    hipcub::StoreDirectStriped<BLOCK_THREADS>(threadIdx.x, d_out + block_offset, items);
}

template<typename T>
__device__ inline T dev_dot(T * arr1, T * arr2) {
    return arr1[0] * arr2[0] + arr1[1] * arr2[1] + arr1[2] * arr2[2];
}

template<typename T>
__device__ inline void spline5_switch(T & vv, T & dd, T & xx, const float & rmin, const float & rmax) 
{
    if (xx < rmin) {
        dd = 0;
        vv = 1;
    }
    else if (xx < rmax) {
        T uu = (xx - rmin) / (rmax - rmin) ;
        T du = 1. / (rmax - rmin) ;
        vv = uu*uu*uu * (-6 * uu*uu + 15 * uu - 10) + 1;
        dd = ( 3 * uu*uu * (-6 * uu*uu + 15 * uu - 10) + uu*uu*uu * (-12 * uu + 15) ) * du;
    }
    else {
        dd = 0;
        vv = 0;
    }
}

__global__ void get_i_idx_se_a(const int nloc, const int * ilist, int * i_idx)
{
    const unsigned int idy = blockIdx.x * blockDim.x + threadIdx.x;
    if(idy >= nloc) {return;}

    i_idx[ilist[idy]] = idy;
}

template<typename T>
__global__ void format_nlist_fill_a_se_a(
    const T * coord,
    const int  * type,
    const int  * jrange,
    const int  * jlist,
    const float rcut,
    int_64 * key,
    int * i_idx,
    const int MAGIC_NUMBER)
{   
    // <<<nloc, MAGIC_NUMBER>>>
    const unsigned int idx = blockIdx.x;
    const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    const int nsize = jrange[i_idx[idx] + 1] - jrange[i_idx[idx]];
    if (idy >= nsize) {return;}

    const int * nei_idx = jlist + jrange[i_idx[idx]];
    // dev_copy(nei_idx, &jlist[jrange[i_idx]], nsize);

    int_64 * key_in = key + idx * MAGIC_NUMBER;

    T diff[3];
    const int & j_idx = nei_idx[idy];
    for (int dd = 0; dd < 3; dd++) {
        diff[dd] = coord[j_idx * 3 + dd] - coord[idx * 3 + dd];
    }
    T rr = sqrt(dev_dot(diff, diff)); 
    if (rr <= rcut) {
        key_in[idy] = (int_64)(type[j_idx] * 1E15) + (int_64)(rr * 1.0E13) / 100000 * 100000 + j_idx;
    }
}

    // bubble_sort(sel_nei, num_nei);
__global__ void format_nlist_fill_b_se_a(
    int * nlist,
    const int nlist_size,
    const int nloc,
    const int * jrange,
    const int * jlist,
    int_64 * key,
    const int * sec_a,
    const int sec_a_size,
    int * nei_iter_dev,
    const int MAGIC_NUMBER)
{ 

    const unsigned int idy = blockIdx.x * blockDim.x + threadIdx.x;

    if(idy >= nloc) {
        return;
    }
    
    int * row_nlist = nlist + idy * nlist_size;
    int * nei_iter = nei_iter_dev + idy * sec_a_size;
    int_64 * key_out = key + nloc * MAGIC_NUMBER + idy * MAGIC_NUMBER;

    for (int ii = 0; ii < sec_a_size; ii++) {
        nei_iter[ii] = sec_a[ii];
    }
    
    for (unsigned int kk = 0; key_out[kk] != key_out[MAGIC_NUMBER - 1]; kk++) {
        const int & nei_type = key_out[kk] / 1E15;
        if (nei_iter[nei_type] < sec_a[nei_type + 1]) {
            row_nlist[nei_iter[nei_type]++] = key_out[kk] % 100000;
        }
    }
}
//it's ok!

template<typename T>
__global__ void compute_descriptor_se_a (
    T* descript,
    const int ndescrpt,
    T* descript_deriv,
    const int descript_deriv_size,
    T* rij,
    const int rij_size,
    const int* type,
    const T* avg,
    const T* std,
    int* nlist,
    const int nlist_size,
    const T* coord,
    const float rmin,
    const float rmax,
    const int sec_a_size)
{   
    // <<<nloc, sec_a.back()>>>
    const unsigned int idx = blockIdx.x;
    const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
    const int idx_deriv = idy * 4 * 3;	// 4 components time 3 directions
    const int idx_value = idy * 4;	// 4 components
    if (idy >= sec_a_size) {return;}

    // else {return;}
    T * row_descript = descript + idx * ndescrpt;
    T * row_descript_deriv = descript_deriv + idx * descript_deriv_size;
    T * row_rij = rij + idx * rij_size;
    int * row_nlist = nlist + idx * nlist_size;

    if (row_nlist[idy] >= 0) {
        const int & j_idx = row_nlist[idy];
        for (int kk = 0; kk < 3; kk++) {
            row_rij[idy * 3 + kk] = coord[j_idx * 3 + kk] - coord[idx * 3 + kk];
        }
        const T * rr = &row_rij[idy * 3 + 0];
        T nr2 = dev_dot(rr, rr);
        T inr = 1./sqrt(nr2);
        T nr = nr2 * inr;
        T inr2 = inr * inr;
        T inr4 = inr2 * inr2;
        T inr3 = inr4 * nr;
        T sw, dsw;
        spline5_switch(sw, dsw, nr, rmin, rmax);
        row_descript[idx_value + 0] = (1./nr)       ;//* sw;
        row_descript[idx_value + 1] = (rr[0] / nr2) ;//* sw;
        row_descript[idx_value + 2] = (rr[1] / nr2) ;//* sw;
        row_descript[idx_value + 3] = (rr[2] / nr2) ;//* sw;

        row_descript_deriv[idx_deriv + 0] = (rr[0] * inr3 * sw - row_descript[idx_value + 0] * dsw * rr[0] * inr); // avg[type[(idx_deriv + 0) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 0) % (ndescrpt * 3)) / 3];
        row_descript_deriv[idx_deriv + 1] = (rr[1] * inr3 * sw - row_descript[idx_value + 0] * dsw * rr[1] * inr); // avg[type[(idx_deriv + 1) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 1) % (ndescrpt * 3)) / 3];
        row_descript_deriv[idx_deriv + 2] = (rr[2] * inr3 * sw - row_descript[idx_value + 0] * dsw * rr[2] * inr); // avg[type[(idx_deriv + 2) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 2) % (ndescrpt * 3)) / 3];
        // ****deriv of component x/r2
        row_descript_deriv[idx_deriv + 3] = ((2. * rr[0] * rr[0] * inr4 - inr2) * sw - row_descript[idx_value + 1] * dsw * rr[0] * inr); // avg[type[(idx_deriv + 3) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 3) % (ndescrpt * 3)) / 3];
        row_descript_deriv[idx_deriv + 4] = ((2. * rr[0] * rr[1] * inr4	) * sw - row_descript[idx_value + 1] * dsw * rr[1] * inr); // avg[type[(idx_deriv + 4) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 4) % (ndescrpt * 3)) / 3];
        row_descript_deriv[idx_deriv + 5] = ((2. * rr[0] * rr[2] * inr4	) * sw - row_descript[idx_value + 1] * dsw * rr[2] * inr); // avg[type[(idx_deriv + 5) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 5) % (ndescrpt * 3)) / 3];
        // ***deriv of component y/r2
        row_descript_deriv[idx_deriv + 6] = ((2. * rr[1] * rr[0] * inr4	) * sw - row_descript[idx_value + 2] * dsw * rr[0] * inr); // avg[type[(idx_deriv + 6) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 6) % (ndescrpt * 3)) / 3];
        row_descript_deriv[idx_deriv + 7] = ((2. * rr[1] * rr[1] * inr4 - inr2) * sw - row_descript[idx_value + 2] * dsw * rr[1] * inr); // avg[type[(idx_deriv + 7) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 7) % (ndescrpt * 3)) / 3];
        row_descript_deriv[idx_deriv + 8] = ((2. * rr[1] * rr[2] * inr4	) * sw - row_descript[idx_value + 2] * dsw * rr[2] * inr); // avg[type[(idx_deriv + 8) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 8) % (ndescrpt * 3)) / 3];
        // ***deriv of component z/r2
        row_descript_deriv[idx_deriv + 9] = ((2. * rr[2] * rr[0] * inr4	) * sw - row_descript[idx_value + 3] * dsw * rr[0] * inr); // avg[type[(idx_deriv + 9) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 9) % (ndescrpt * 3)) / 3];
        row_descript_deriv[idx_deriv +10] = ((2. * rr[2] * rr[1] * inr4	) * sw - row_descript[idx_value + 3] * dsw * rr[1] * inr); // avg[type[(idx_deriv + 10) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 10) % (ndescrpt * 3)) / 3];
        row_descript_deriv[idx_deriv +11] = ((2. * rr[2] * rr[2] * inr4 - inr2) * sw - row_descript[idx_value + 3] * dsw * rr[2] * inr); // avg[type[(idx_deriv + 11) / (ndescrpt * 3)] * ndescrpt + ((idx_deriv + 11) % (ndescrpt * 3)) / 3];
        // 4 value components
        row_descript[idx_value + 0] *= sw; // * descript[idx * ndescrpt + idx_value + 0]);// - avg[type[idx] * ndescrpt + idx_value + 0]) / std[type[idx] * ndescrpt + idx_value + 0];
        row_descript[idx_value + 1] *= sw; // * descript[idx * ndescrpt + idx_value + 1]);// - avg[type[idx] * ndescrpt + idx_value + 1]) / std[type[idx] * ndescrpt + idx_value + 1];
        row_descript[idx_value + 2] *= sw; // * descript[idx * ndescrpt + idx_value + 2]);// - avg[type[idx] * ndescrpt + idx_value + 2]) / std[type[idx] * ndescrpt + idx_value + 2];
        row_descript[idx_value + 3] *= sw; // * descript[idx * ndescrpt + idx_value + 3]);// - avg[type[idx] * ndescrpt + idx_value + 3]) / std[type[idx] * ndescrpt + idx_value + 3];
    }

    for (int ii = 0; ii < 4; ii++) {
        row_descript[idx_value + ii] = (row_descript[idx_value + ii] - avg[type[idx] * ndescrpt + idx_value + ii]) / std[type[idx] * ndescrpt + idx_value + ii];
    }
    for (int ii = 0; ii < 12; ii++) {
        row_descript_deriv[idx_deriv + ii] /= std[type[idx] * ndescrpt + (idx_deriv + ii) / 3];
    }
}

template<typename T>
void format_nbor_list_256 (const T* coord, const int* type, const int* jrange, const int* jlist, const int& nloc, const float& rcut_r, int * i_idx, int_64 * key)  {   
    const int LEN = 256;
    const int MAGIC_NUMBER = 256;
    const int nblock = (MAGIC_NUMBER + LEN - 1) / LEN;
    dim3 block_grid(nloc, nblock);
    dim3 thread_grid(1, LEN);
    format_nlist_fill_a_se_a<<<block_grid, thread_grid>>> (coord, type, jrange, jlist, rcut_r, key, i_idx, MAGIC_NUMBER);
    
    const int ITEMS_PER_THREAD = 4;
    const int BLOCK_THREADS = MAGIC_NUMBER / ITEMS_PER_THREAD;
    BlockSortKernel<int_64, BLOCK_THREADS, ITEMS_PER_THREAD> <<<nloc, BLOCK_THREADS>>> (key, key + nloc * MAGIC_NUMBER);
}

template<typename T>
void format_nbor_list_512 (const T* coord, const int* type, const int* jrange, const int* jlist, const int& nloc, const float& rcut_r, int * i_idx, int_64 * key)  {   
    const int LEN = 256;
    const int MAGIC_NUMBER = 512;
    const int nblock = (MAGIC_NUMBER + LEN - 1) / LEN;
    dim3 block_grid(nloc, nblock);
    dim3 thread_grid(1, LEN);
    format_nlist_fill_a_se_a<<<block_grid, thread_grid>>> (coord, type, jrange, jlist, rcut_r, key, i_idx, MAGIC_NUMBER);

    const int ITEMS_PER_THREAD = 4;
    const int BLOCK_THREADS = MAGIC_NUMBER / ITEMS_PER_THREAD;
    BlockSortKernel<int_64, BLOCK_THREADS, ITEMS_PER_THREAD> <<<nloc, BLOCK_THREADS>>> (key, key + nloc * MAGIC_NUMBER);
}

template<typename T>
void format_nbor_list_1024 (const T* coord, const int* type, const int* jrange, const int* jlist, const int& nloc, const float& rcut_r, int * i_idx, int_64 * key)  {   
    const int LEN = 256;
    const int MAGIC_NUMBER = 1024;
    const int nblock = (MAGIC_NUMBER + LEN - 1) / LEN;
    dim3 block_grid(nloc, nblock);
    dim3 thread_grid(1, LEN);
    format_nlist_fill_a_se_a<<<block_grid, thread_grid>>> (coord, type, jrange, jlist, rcut_r, key, i_idx, MAGIC_NUMBER);

    const int ITEMS_PER_THREAD = 8;
    const int BLOCK_THREADS = MAGIC_NUMBER / ITEMS_PER_THREAD;
    BlockSortKernel<int_64, BLOCK_THREADS, ITEMS_PER_THREAD> <<<nloc, BLOCK_THREADS>>> (key, key + nloc * MAGIC_NUMBER);
}

template<typename T>
void format_nbor_list_2048 (const T* coord, const int* type, const int* jrange, const int* jlist, const int& nloc, const float& rcut_r, int * i_idx, int_64 * key)  {   
    const int LEN = 256;
    const int MAGIC_NUMBER = 2048;
    const int nblock = (MAGIC_NUMBER + LEN - 1) / LEN;
    dim3 block_grid(nloc, nblock);
    dim3 thread_grid(1, LEN);
    format_nlist_fill_a_se_a<<<block_grid, thread_grid>>> (coord, type, jrange, jlist, rcut_r, key, i_idx, MAGIC_NUMBER);

    const int ITEMS_PER_THREAD = 8;
    const int BLOCK_THREADS = MAGIC_NUMBER / ITEMS_PER_THREAD;
    BlockSortKernel<int_64, BLOCK_THREADS, ITEMS_PER_THREAD> <<<nloc, BLOCK_THREADS>>> (key, key + nloc * MAGIC_NUMBER);
}

template<typename T>
void format_nbor_list_4096 (const T* coord, const int* type, const int* jrange, const int* jlist, const int& nloc, const float& rcut_r, int * i_idx, int_64 * key)  {   
    const int LEN = 256;
    const int MAGIC_NUMBER = 4096;
    const int nblock = (MAGIC_NUMBER + LEN - 1) / LEN;
    dim3 block_grid(nloc, nblock);
    dim3 thread_grid(1, LEN);
    format_nlist_fill_a_se_a<<<block_grid, thread_grid>>> (coord, type, jrange, jlist, rcut_r, key, i_idx, MAGIC_NUMBER);

    const int ITEMS_PER_THREAD = 16;
    const int BLOCK_THREADS = MAGIC_NUMBER / ITEMS_PER_THREAD;
    BlockSortKernel<int_64, BLOCK_THREADS, ITEMS_PER_THREAD> <<<nloc, BLOCK_THREADS>>> (key, key + nloc * MAGIC_NUMBER);
}

void DescrptSeAGPUExecuteLauncher(const float * coord, const int * type, const int * ilist, const int * jrange, const int * jlist, int * array_int, unsigned long long * array_longlong, const float * avg, const float * std, float * descript, float * descript_deriv, float * rij, int * nlist, const int nloc, const int nall, const int nnei, const int ndescrpt, const float rcut_r, const float rcut_r_smth, const std::vector<int> sec_a, const bool fill_nei_a, const int MAGIC_NUMBER)
{   
    const int LEN = 256;
    int nblock = (nloc + LEN -1) / LEN;
    int * sec_a_dev = array_int;
    int * nei_iter = array_int + sec_a.size(); // = new int[sec_a_size];
    int * i_idx = array_int + sec_a.size() + nloc * sec_a.size();
    int_64 * key = array_longlong;
    
    hipError_t res = hipSuccess;
    res = hipMemcpy(sec_a_dev, &sec_a[0], sizeof(int) * sec_a.size(), hipMemcpyHostToDevice); cudaErrcheck(res);    
    res = hipMemset(key, 0xffffffff, sizeof(int_64) * nloc * MAGIC_NUMBER); cudaErrcheck(res);
    res = hipMemset(nlist, -1, sizeof(int) * nloc * nnei); cudaErrcheck(res);
    res = hipMemset(descript, 0.0, sizeof(float) * nloc * ndescrpt); cudaErrcheck(res);
    res = hipMemset(descript_deriv, 0.0, sizeof(float) * nloc * ndescrpt * 3); cudaErrcheck(res);

    if (fill_nei_a) {
        // ~~~
        // hipProfilerStart();
        get_i_idx_se_a<<<nblock, LEN>>> (nloc, ilist, i_idx);

        if (nnei <= 256) {
            format_nbor_list_256 (coord, type, jrange, jlist, nloc, rcut_r, i_idx, key); 
        } else if (nnei <= 512) {
            format_nbor_list_512 (coord, type, jrange, jlist, nloc, rcut_r, i_idx, key); 
        } else if (nnei <= 1024) {
            format_nbor_list_1024 (coord, type, jrange, jlist, nloc, rcut_r, i_idx, key); 
        } else if (nnei <= 2048) {
            format_nbor_list_2048 (coord, type, jrange, jlist, nloc, rcut_r, i_idx, key); 
        } else if (nnei <= 4096) {
            format_nbor_list_4096 (coord, type, jrange, jlist, nloc, rcut_r, i_idx, key); 
        } 

        format_nlist_fill_b_se_a<<<nblock, LEN>>> (nlist, nnei, nloc, jrange, jlist, key, sec_a_dev, sec_a.size(), nei_iter, MAGIC_NUMBER);
    }

    const int nblock_ = (sec_a.back() + LEN -1) / LEN;
    dim3 block_grid(nloc, nblock_);
    dim3 thread_grid(1, LEN);
    compute_descriptor_se_a<<<block_grid, thread_grid>>> (descript, ndescrpt, descript_deriv, ndescrpt * 3, rij, nnei * 3, type, avg, std, nlist, nnei, coord, rcut_r_smth, rcut_r, sec_a.back());
}

void DescrptSeAGPUExecuteLauncher(const double * coord, const int * type, const int * ilist, const int * jrange, const int * jlist, int * array_int, unsigned long long * array_longlong, const double * avg, const double * std, double * descript, double * descript_deriv, double * rij, int * nlist, const int nloc, const int nall, const int nnei, const int ndescrpt, const float rcut_r, const float rcut_r_smth, const std::vector<int> sec_a, const bool fill_nei_a, const int MAGIC_NUMBER)
{   
    const int LEN = 256;
    int nblock = (nloc + LEN -1) / LEN;
    int * sec_a_dev = array_int;
    int * nei_iter = array_int + sec_a.size(); // = new int[sec_a_size];
    int * i_idx = array_int + sec_a.size() + nloc * sec_a.size();
    int_64 * key = array_longlong;
    
    hipError_t res = hipSuccess;
    res = hipMemcpy(sec_a_dev, &sec_a[0], sizeof(int) * sec_a.size(), hipMemcpyHostToDevice); cudaErrcheck(res);    
    res = hipMemset(key, 0xffffffff, sizeof(int_64) * nloc * MAGIC_NUMBER); cudaErrcheck(res);
    res = hipMemset(nlist, -1, sizeof(int) * nloc * nnei); cudaErrcheck(res);
    res = hipMemset(descript, 0.0, sizeof(double) * nloc * ndescrpt); cudaErrcheck(res);
    res = hipMemset(descript_deriv, 0.0, sizeof(double) * nloc * ndescrpt * 3); cudaErrcheck(res);

    if (fill_nei_a) {
        // ~~~
        // hipProfilerStart();
        get_i_idx_se_a<<<nblock, LEN>>> (nloc, ilist, i_idx);

        if (nnei <= 256) {
            format_nbor_list_256 (coord, type, jrange, jlist, nloc, rcut_r, i_idx, key); 
        }  else if (nnei <= 512) {
            format_nbor_list_512 (coord, type, jrange, jlist, nloc, rcut_r, i_idx, key); 
        } else if (nnei <= 1024) {
            format_nbor_list_1024 (coord, type, jrange, jlist, nloc, rcut_r, i_idx, key); 
        } else if (nnei <= 2048) {
            format_nbor_list_2048 (coord, type, jrange, jlist, nloc, rcut_r, i_idx, key); 
        } else if (nnei <= 4096) {
            format_nbor_list_4096 (coord, type, jrange, jlist, nloc, rcut_r, i_idx, key); 
        }

        format_nlist_fill_b_se_a<<<nblock, LEN>>> (nlist, nnei, nloc, jrange, jlist, key, sec_a_dev, sec_a.size(), nei_iter, MAGIC_NUMBER);
    }

    const int nblock_ = (sec_a.back() + LEN -1) / LEN;
    dim3 block_grid(nloc, nblock_);
    dim3 thread_grid(1, LEN);
    compute_descriptor_se_a<<<block_grid, thread_grid>>> (descript, ndescrpt, descript_deriv, ndescrpt * 3, rij, nnei * 3, type, avg, std, nlist, nnei, coord, rcut_r_smth, rcut_r, sec_a.back());

    cudaErrcheck(hipGetLastError());

    #if GPU_DEBUG
    #include<fstream>
    std::fstream fout1("nlist_gpu.txt", std::ios::out);
    int * nlist_host = (int *)malloc(sizeof(int) * nloc * nnei);
    cudaErrcheck(hipMemcpy(nlist_host, nlist, sizeof(int) * nloc * nnei, hipMemcpyDeviceToHost));
    fout1 << "nlist info:\t" << "height:\t" << nloc << "\twidth:\t" << nnei << std::endl;
    for (int ii = 0; ii < nloc; ii++) {
        for (int jj = 0; jj < nnei; jj++) {
            fout1 << nlist_host[ii * nnei + jj] << "\t";
        }
        fout1 << std::endl;
    }
    fout1.close();

    std::fstream fout2("rij_gpu.txt", std::ios::out);
    double * rij_host = (double *)malloc(sizeof(double) * nloc * nnei * 3);
    cudaErrcheck(hipMemcpy(rij_host, rij, sizeof(double) * nloc * nnei * 3, hipMemcpyDeviceToHost));
    fout2 << "rij info:\t" << "height:\t" << nloc << "\twidth:\t" << nnei * 3 << std::endl;
    for (int ii = 0; ii < nloc; ii++) {
        for (int jj = 0; jj < nnei * 3; jj++) {
            fout2 << rij_host[ii * nnei * 3 + jj] << "\t";
        }
        fout2 << std::endl;
    }
    fout2.close();

    std::fstream fout3("descrpt_gpu.txt", std::ios::out);
    double * descrpt_host = (double *)malloc(sizeof(double) * nloc * ndescrpt);
    cudaErrcheck(hipMemcpy(descrpt_host, descript, sizeof(double) * nloc * ndescrpt, hipMemcpyDeviceToHost));
    fout3 << "descrpt info:\t" << "height:\t" << nloc << "\twidth:\t" << ndescrpt << std::endl;
    for (int ii = 0; ii < nloc; ii++) {
        for (int jj = 0; jj < ndescrpt; jj++) {
            fout3 << descrpt_host[ii * ndescrpt + jj] << "\t";
        }
        fout3 << std::endl;
    }
    fout3.close();


    std::fstream fout4("descrpt_deriv_gpu.txt", std::ios::out);
    double * descrpt_deriv_host = (double *)malloc(sizeof(double) * nloc * ndescrpt * 3);
    cudaErrcheck(hipMemcpy(descrpt_deriv_host, descript_deriv, sizeof(double) * nloc * ndescrpt * 3, hipMemcpyDeviceToHost));
    fout4 << "descrpt_deriv info:\t" << "height:\t" << nloc << "\twidth:\t" << ndescrpt * 3 << std::endl;
    for (int ii = 0; ii < nloc; ii++) {
        for (int jj = 0; jj < ndescrpt * 3; jj++) {
            fout4 << descrpt_deriv_host[ii * ndescrpt * 3 + jj] << "\t";
        }
        fout4 << std::endl;
    }
    fout4.close();
    #endif // GPU_DEBUG
}