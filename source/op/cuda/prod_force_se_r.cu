#include <stdio.h>
#include <hip/hip_runtime.h>

#ifdef HIGH_PREC
    typedef double VALUETYPE;
#else
    typedef float  VALUETYPE;
#endif

#define cudaErrcheck(res) { cudaAssert((res), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"cuda assert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
static __inline__ __device__ double atomicAdd(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val + __longlong_as_double(assumed)));
    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN) } while (assumed != old);
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif

__global__ void deriv_wrt_center_atom_se_r(VALUETYPE * force, 
                        const VALUETYPE * net_deriv,
                        const VALUETYPE * in_deriv,
                        const int ndescrpt)
{
    const unsigned int idx = blockIdx.x;
    const unsigned int idy = threadIdx.x;
    const unsigned int idz = blockIdx.y;
    
    atomicAdd(force + idx * 3 + idz, -1.0 * net_deriv[idx * ndescrpt + idy] * in_deriv[idx * ndescrpt * 3 + idy * 3 + idz]);
}

__global__ void deriv_wrt_neighbors_se_r(VALUETYPE * force, 
                        const VALUETYPE * net_deriv,
                        const VALUETYPE * in_deriv,
                        const int * nlist,
                        const int nloc,
                        const int nnei,
                        const int ndescrpt,
                        const int n_a_sel,
                        const int n_a_shift)
{  
    // idy -> nnei
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int idy = blockIdx.y;
    const unsigned int idz = threadIdx.y;
    
    if (idx >= nloc) {
        return;
    }
    // deriv wrt neighbors
    int j_idx = nlist[idx * nnei + idy];
    if (j_idx < 0) {
        return;
    }
    atomicAdd(force + j_idx * 3 + idz, net_deriv[idx * ndescrpt + idy] * in_deriv[idx * ndescrpt * 3 + idy * 3 + idz]);
}

void ProdForceSeRLauncher(VALUETYPE * force, 
                        const VALUETYPE * net_deriv,
                        const VALUETYPE * in_deriv,
                        const int * nlist,
                        const int nloc,
                        const int nall,
                        const int ndescrpt,
                        const int nnei,
                        const int n_a_sel,
                        const int n_a_shift)
{
    cudaErrcheck(hipMemset(force, 0.0, sizeof(VALUETYPE) * nall * 3));
    dim3 grid(nloc, 3);
    deriv_wrt_center_atom_se_r<<<grid, ndescrpt>>>(force, net_deriv, in_deriv, ndescrpt);
    
    const int LEN = 64;
    int nblock = (nloc + LEN -1) / LEN;
    dim3 block_grid(nblock, nnei);
    dim3 thread_grid(LEN, 3);
    deriv_wrt_neighbors_se_r<<<block_grid, thread_grid>>>(force, net_deriv, in_deriv, nlist, nloc, nnei, ndescrpt, n_a_sel, n_a_shift);
}