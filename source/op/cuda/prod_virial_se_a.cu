#include <stdio.h>
#include <hip/hip_runtime.h>

#define MUL 512

#ifdef HIGH_PREC
    typedef double VALUETYPE;
#else
    typedef float  VALUETYPE;
#endif

#define cudaErrcheck(res) { cudaAssert((res), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"cuda assert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
static __inline__ __device__ double atomicAdd(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val + __longlong_as_double(assumed)));
    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN) } while (assumed != old);
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif

__global__ void deriv_wrt_neighbors_se_a(VALUETYPE * virial, 
                        VALUETYPE * atom_virial,
                        const VALUETYPE * net_deriv,
                        const VALUETYPE * in_deriv,
                        const VALUETYPE * rij,
                        const int * nlist,
                        const int nloc,
                        const int nnei,
                        const int ndescrpt,
                        const int n_a_sel,
                        const int n_a_shift) 
{
    // idx -> nloc
    // idy -> nnei
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int idy = blockIdx.y;
    const unsigned int idz = threadIdx.y;
    const unsigned int idw = threadIdx.z;

    if (idx >= nloc) {
        return;
    }
    int j_idx = nlist[idx * nnei + idy];
    if (j_idx < 0) {
        return;
    }
    // atomicAdd(virial + idz, net_deriv[idx * ndescrpt + idy * 4 + idw] * rij[idx * nnei * 3 + idy * 3 + idz / 3] * in_deriv[idx * ndescrpt * 3 + (idy * 4 + idw) * 3 + idz % 3]);
    atomicAdd(atom_virial + j_idx * 9 + idz, net_deriv[idx * ndescrpt + idy * 4 + idw] * rij[idx * nnei * 3 + idy * 3 + idz / 3] * in_deriv[idx * ndescrpt * 3 + (idy * 4 + idw) * 3 + idz % 3]);
}

void ProdVirialSeALauncher(VALUETYPE * virial, 
                        VALUETYPE * atom_virial,
                        const VALUETYPE * net_deriv,
                        const VALUETYPE * in_deriv,
                        const VALUETYPE * rij,
                        const int * nlist,
                        const int nloc,
                        const int nall,
                        const int nnei,
                        const int ndescrpt,
                        const int n_a_sel,
                        const int n_a_shift) 
{
    cudaErrcheck(hipMemset(virial, 0.0, sizeof(VALUETYPE) * 9));
    cudaErrcheck(hipMemset(atom_virial, 0.0, sizeof(VALUETYPE) * 9 * nall));

    const int LEN = 16;
    int nblock = (nloc + LEN -1) / LEN;
    dim3 block_grid(nblock, nnei);
    dim3 thread_grid(LEN, 9, 4);
    // compute virial of a frame
    deriv_wrt_neighbors_se_a<<<block_grid, thread_grid>>>(
                        virial, 
                        atom_virial, 
                        net_deriv, 
                        in_deriv,
                        rij,
                        nlist,
                        nloc,
                        nnei,
                        ndescrpt,
                        n_a_sel,
                        n_a_shift
    );
}
