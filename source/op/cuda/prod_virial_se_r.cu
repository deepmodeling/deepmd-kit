#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>

#define MUL 512

#ifdef HIGH_PREC
    typedef double VALUETYPE;
#else
    typedef float  VALUETYPE;
#endif

#define cudaErrcheck(res) { cudaAssert((res), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"cuda assert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// currently, double precision atomicAdd only support arch number larger than 6.0
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 600
static __inline__ __device__ double atomicAdd(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val + __longlong_as_double(assumed)));
    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN) } while (assumed != old);
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif

__global__ void deriv_wrt_neighbors_se_r(VALUETYPE * virial, 
                        VALUETYPE * atom_virial,
                        const VALUETYPE * net_deriv,
                        const VALUETYPE * in_deriv,
                        const VALUETYPE * rij,
                        const int * nlist,
                        const int nloc,
                        const int nnei,
                        const int ndescrpt,
                        const int n_a_sel,
                        const int n_a_shift) 
{
    // idx -> nloc
    // idy -> nnei
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int idy = blockIdx.y;
    const unsigned int idz = threadIdx.y;

    if (idx >= nloc) {
        return;
    }
    int j_idx = nlist[idx * nnei + idy];
    if (j_idx < 0) {
        return;
    }
    atomicAdd(atom_virial + j_idx * 9 + idz, net_deriv[idx * ndescrpt + idy] * rij[idx * nnei * 3 + idy * 3 + idz / 3] * in_deriv[idx * ndescrpt * 3 + idy * 3 + idz % 3]);
}

void ProdVirialSeRLauncher(VALUETYPE * virial, 
                        VALUETYPE * atom_virial,
                        const VALUETYPE * net_deriv,
                        const VALUETYPE * in_deriv,
                        const VALUETYPE * rij,
                        const int * nlist,
                        const int nloc,
                        const int nall,
                        const int nnei,
                        const int ndescrpt,
                        const int n_a_sel,
                        const int n_a_shift) 
{
    cudaErrcheck(hipMemset(virial, 0.0, sizeof(VALUETYPE) * 9));
    cudaErrcheck(hipMemset(atom_virial, 0.0, sizeof(VALUETYPE) * 9 * nall));

    const int LEN = 16;
    int nblock = (nloc + LEN -1) / LEN;
    dim3 block_grid(nblock, nnei);
    dim3 thread_grid(LEN, 9);
    // compute virial of a frame
    deriv_wrt_neighbors_se_r<<<block_grid, thread_grid>>>(
                        virial, 
                        atom_virial, 
                        net_deriv, 
                        in_deriv,
                        rij,
                        nlist,
                        nloc,
                        nnei,
                        ndescrpt,
                        n_a_sel,
                        n_a_shift
    );
}